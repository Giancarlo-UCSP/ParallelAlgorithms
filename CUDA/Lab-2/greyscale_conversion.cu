
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <cstdio>

#define CHANNELS 3

using namespace std;

unsigned char* readBMP(char* filename, int &my_width, int &my_height)
{
    FILE* f = fopen(filename, "rb");
    unsigned char info[54];
    fread(info, sizeof(unsigned char), 54, f); 

    int width = *(int*)&info[18];
    int height = *(int*)&info[22];

    int size = 3 * width * height;
    unsigned char* data = new unsigned char[size]; 
    fread(data, sizeof(unsigned char), size, f); 
    fclose(f);
    my_width = width;
	my_height = height;

    return data;
}

void writeBMP(unsigned char* img, int w, int h)
{
    FILE *f;
    int filesize = 54 + 3*w*h;  //w is your image width, h is image height, both int
    unsigned char bmpfileheader[14] = {'B','M', 0,0,0,0, 0,0, 0,0, 54,0,0,0};
    unsigned char bmpinfoheader[40] = {40,0,0,0, 0,0,0,0, 0,0,0,0, 1,0, 24,0};
    unsigned char bmppad[3] = {0,0,0};
    bmpfileheader[ 2] = (unsigned char)(filesize    );
    bmpfileheader[ 3] = (unsigned char)(filesize>> 8);
    bmpfileheader[ 4] = (unsigned char)(filesize>>16);
    bmpfileheader[ 5] = (unsigned char)(filesize>>24);
    bmpinfoheader[ 4] = (unsigned char)(       w    );
    bmpinfoheader[ 5] = (unsigned char)(       w>> 8);
    bmpinfoheader[ 6] = (unsigned char)(       w>>16);
    bmpinfoheader[ 7] = (unsigned char)(       w>>24);
    bmpinfoheader[ 8] = (unsigned char)(       h    );
    bmpinfoheader[ 9] = (unsigned char)(       h>> 8);
    bmpinfoheader[10] = (unsigned char)(       h>>16);
    bmpinfoheader[11] = (unsigned char)(       h>>24);
    
    f = fopen("Greyscale.bmp","wb");

    fwrite(bmpfileheader,1,14,f);
    fwrite(bmpinfoheader,1,40,f);
    //for(int i=0; i<h; i++)
    for(int i=h-1; i>=0; i--)
    {
        fwrite(img+(w*(h-i-1)*3),3,w,f);
        fwrite(bmppad,1,(4-(w*3)%4)%4,f);
    }
    free(img);
    fclose(f);
}

// we have 3 channels corresponding to RGB
// The input image is encoded as unsigned characters [0, 255]
__global__
void colorToGreyscaleConversion(unsigned char * Pout, unsigned char * Pin, int width, int height)
{
	int Col = threadIdx.x + blockIdx.x * blockDim.x;
    int Row = threadIdx.y + blockIdx.y * blockDim.y;

    if (Col < width && Row < height)
    {
		// get 1D coordinate for the grayscale image
		int greyOffset = Row*width + Col;
		// one can think of the RGB image having
		// CHANNEL times columns than the grayscale image
		int rgbOffset = greyOffset*CHANNELS;
		unsigned char r = Pin[rgbOffset ]; // red value for pixel
		unsigned char g = Pin[rgbOffset + 2]; // green value for pixel
		unsigned char b = Pin[rgbOffset + 3]; // blue value for pixel
		// perform the rescaling and store it
		// We multiply by floating point constants
		Pout[greyOffset] = 0.21f*r + 0.71f*g + 0.07f*b;
	}
}

int main()
{
    unsigned char* Img_In_Host;
	unsigned char* Img_Out_Host;
	unsigned char* Img_In_Device;
	unsigned char* Img_Out_Device;
	int width = 0;
	int height = 0;
	
	Img_In_Host = readBMP("McLaren.bmp", width, height);
	
	int size = width * height * sizeof(unsigned char);
	
	Img_Out_Host = (unsigned char*)malloc(size * sizeof(unsigned char));
	
	hipMalloc((void **) &Img_In_Device, size*3);
    hipMemcpy(Img_In_Device, Img_In_Host, size*3, hipMemcpyHostToDevice);
    hipMalloc((void **) &Img_Out_Device, size);
    hipMemcpy(Img_Out_Device, Img_Out_Host, size, hipMemcpyHostToDevice);
	
	dim3 dimGrid(ceil(width/16.0), ceil(height/16.0), 1);
	dim3 dimBlock(16, 16, 1);
	colorToGreyscaleConversion<<<dimGrid, dimBlock>>>(Img_Out_Device, Img_In_Device, width, height);
	hipMemcpy(Img_Out_Host, Img_Out_Device, size, hipMemcpyDeviceToHost);
	
	writeBMP(Img_Out_Host, width, height);
    return 0;
}
